#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>


__device__ float doubl_d(float in)
{
  return in * 2;
}

__global__ void doubl_g(float* in, float* out)
{
  int idx = threadIdx.x;
  out[idx] = doubl_d(in[idx]);
}

extern "C"
__host__ void doubl_h(float* in, float* out)
{
  const int SIZE = 64;
  size_t BYTES = SIZE * sizeof(float);

  float* h_in = in;
  float* h_out = out;

  float* d_in;
  float* d_out;
  hipMalloc((void **) &d_in, BYTES);
  hipMalloc((void **) &d_out, BYTES);

  hipMemcpy(d_in, h_in, BYTES, hipMemcpyHostToDevice);

  doubl_g<<<1,SIZE>>>(d_in, d_out);

  hipMemcpy(h_out, d_out, BYTES, hipMemcpyDeviceToHost);

}