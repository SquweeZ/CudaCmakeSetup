
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

__device__ float square_d(float in)
{
  return in * in;
}

__global__ void square_g(float *d_in, float *d_out)
{

  int idx = threadIdx.x;
  d_out[idx] = square_d(d_in[idx]);
}


extern "C"
void squareIt_h(float *in, float *out)
{
  const int SIZE = 64;
  const size_t BYTES = SIZE * sizeof(float);
  float* h_in = in;

  float* h_out = out;

  float* d_in;
  float* d_out;

  hipMalloc((void **) &d_in, BYTES);
  hipMalloc((void **) &d_out, BYTES);

  hipMemcpy(d_in, h_in, BYTES, hipMemcpyHostToDevice);

  square_g<<<1, SIZE>>>(d_in, d_out);

  hipMemcpy(h_out, d_out, BYTES, hipMemcpyDeviceToHost);

  return ;
}